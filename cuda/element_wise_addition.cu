// simple cuda element-wise addition
// requires nvcc to compile and run


#include <hip/hip_runtime.h>
#include <stdio.h>


__global__ void add(int *a, int *b, int *c, int n) {
    int index = threadIdx.x;
    if (index < n) {
        c[index] = a[index] + b[index];
    }
}

int main() {
    const int n = 5;
    int h_a[n] = {1, 2, 3, 4, 5};   // Host input array a
    int h_b[n] = {10, 20, 30, 40, 50}; // Host input array b
    int h_c[n]; // Host output array c

    int *d_a, *d_b, *d_c; // Device pointers

    // Allocate device memory
    hipMalloc((void**)&d_a, n * sizeof(int));
    hipMalloc((void**)&d_b, n * sizeof(int));
    hipMalloc((void**)&d_c, n * sizeof(int));

    // Copy inputs to device
    hipMemcpy(d_a, h_a, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, n * sizeof(int), hipMemcpyHostToDevice);

    // Launch the kernel on the GPU
    add<<<1, n>>>(d_a, d_b, d_c, n);

    // Copy result back to host
    hipMemcpy(h_c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);

    // Print the result
    printf("Result:\n");
    for (int i = 0; i < n; i++) {
        printf("%d + %d = %d\n", h_a[i], h_b[i], h_c[i]);
    }

    // Free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}